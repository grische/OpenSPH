#include "hip/hip_runtime.h"
#include "DeviceMath.h"
#include <assert.h>
#include <stdio.h>
#include <vector>

using namespace Cuda;

__global__ void evalKernel(float3* r, float* m, float3* dv, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n) {
        return;
    }
    for (int j = 0; j < n; ++j) {
        if (i == j) {
            continue;
        }
        const float3 dr = r[i] - r[j];
        const float x = getLength(dr);
        dv[i] = dv[i] - m[j] * dr / (x * x * x);
    }
}

extern hipError_t cudaEvalAll(float3* r, float* m, float3* dv, int n) {

    // printf("cudaEvalAll called with %#016x\n", m);
    // printf("device m = %d\n", m[0]);

    /*printf("cudaEvalAll called with %#016x\n", m);

    printf("N = %d\n", n);
    printf("m0 = %d\n", m[0]);

    printf("size of pointer = %d\n", sizeof(r));


    printf("r0 = %d\n", r[0]);*/

    /*for (int i = 0; i < 3 * n; ++i) {
        printf("INPUT r = %d\n", r[i]);
    }

    for (int i = 0; i < n; ++i) {
        printf("INPUT m = %d\n", m[i]);
    }*/


    const int blockSize = 256;
    const int numBlocks = (n + blockSize - 1) / blockSize;

    evalKernel<<<numBlocks, blockSize>>>(r, m, dv, n);
    // evalKernel<<<numBlocks, blockSize>>>(r, m, dv, n);

    //    hipDeviceSynchronize();

    return hipSuccess;
}
